#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <memory>
#include <sstream>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"
#include ""
#include "files.h"
#include "timer.h"

constexpr float SOFTENING = 1e-9f;

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */
struct Body {
  float x, y, z;     // position
  float vx, vy, vz;  // speed
};

/*
 * CUDA kernel to compute gravitational forces between all bodies.
 */
__global__ void bodyForce(Body* p, float dt, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread index
  int stride = blockDim.x * gridDim.x;                // Total number of threads

  for (int i = index; i < n; i += stride) {
    float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

    for (int j = 0; j < n; ++j) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3;
      Fy += dy * invDist3;
      Fz += dz * invDist3;
    }

    // Update velocity based on force
    p[i].vx += dt * Fx;
    p[i].vy += dt * Fy;
    p[i].vz += dt * Fz;
  }
}

int main(int argc, char** argv) {
  int nBodies = 1 << 12;  // means 2 * 2^11 = 4096
  if (argc > 1) {
    try {
      nBodies = 2 << std::stoi(argv[1]);  // means 2 * 2^argv[1]
    } catch (...) {
      std::cerr << "Invalid argument for body count. Using default.\n";
    }
  }

  std::string base_path = "files/";
  std::string file_suffix = (nBodies == (1 << 12)) ? "4096" : "65536";
  std::string initialized_values = base_path + "initialized_" + file_suffix;
  std::string solution_values = base_path + "solution_" + file_suffix;

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  constexpr float dt = 0.01f;
  constexpr int nIters = 10;

  // Set active GPU device
  if (hipSetDevice(0) != hipSuccess) {
    std::cerr << "Failed to set CUDA device.\n";
    return 1;
  }
  int deviceId;
  hipGetDevice(&deviceId);  // Get current GPU device ID

  size_t size = static_cast<size_t>(nBodies) * sizeof(Body);
  size_t nFloats = size / sizeof(float);
  Body* p = nullptr;

  // Allocate unified memory accessible by CPU and GPU
  if (hipMallocManaged(&p, size) != hipSuccess) {
    std::cerr << "CUDA malloc error.\n";
    return 1;
  }

  // Prefetch memory to GPU to improve performance
  hipMemPrefetchAsync(p, size, deviceId);

  // Read initial values into a temporary float vector
  std::vector<float> inputBuffer(nFloats);
  read_values_from_file(initialized_values, inputBuffer.data(), nFloats);
  std::memcpy(reinterpret_cast<float*>(p), inputBuffer.data(), size);

  int threadsPerBlock = 256;
  int numberOfBlocks = (nBodies + threadsPerBlock - 1) / threadsPerBlock;

  hipStream_t stream;
  hipStreamCreate(&stream);  // Create CUDA stream for concurrent operations

  double totalTime = 0.0;
  for (int iter = 0; iter < nIters; ++iter) {
    StartTimer();

    // Launch kernel on the specified CUDA stream
    bodyForce<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(p, dt, nBodies);

    // Synchronize to wait for kernel execution to finish
    hipStreamSynchronize(stream);

    // Integrate position based on updated velocity
    for (int i = 0; i < nBodies; ++i) {
      p[i].x += p[i].vx * dt;
      p[i].y += p[i].vy * dt;
      p[i].z += p[i].vz * dt;
    }

    totalTime += GetTimer() / 1000.0;
  }

  // Destroy the CUDA stream after use
  hipStreamDestroy(stream);

  double avgTime = totalTime / static_cast<double>(nIters);
  double interactionsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  std::ostringstream interactionsPerSecondString;
  interactionsPerSecondString << std::fixed << std::setprecision(3)
                              << interactionsPerSecond;
  std::cout << interactionsPerSecondString.str()
            << " Billion Interactions / second\n";

  // Write output data to file
  std::vector<float> outputBuffer(reinterpret_cast<float*>(p),
                                  reinterpret_cast<float*>(p) + nFloats);
  write_values_to_file(solution_values, outputBuffer.data(), nFloats);

  // Free unified memory
  hipFree(p);

  return 0;
}
